#include "hip/hip_runtime.h"
//
// standard header files
//

#include <stdio.h>
#include <stdlib.h>

//
// CUDA header file
//

#include <hip/hip_runtime.h>

//
// my header file
//

#include "poissinv_cuda.h"

__global__ void poissinvf_test(int N, float lam) {

  float x, u;
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;

  u = (tid + 0.5f) / N;

  for (int n=0; n<10; n++) {
    u += 1e-30f;

    if (lam>800.0f)
      x = normcdfinvf(u);
    else {
      if (lam>400.0f) {
        int n = 1 << (tid & 7); 
        lam   = 1.0f * (float) n;
      }
      x = poissinvf(u, lam);
    }

// needed to prevent compiler discarding everything
    if (x==-999.0f) printf("negative x\n");
  }
}

__global__ void poissinv_test(int N, float lam) {

  float x, u;
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;

  u = (tid + 0.5f) / N;

  for (int n=0; n<10; n++) {
    u += 1e-30f;

    if (lam>800.0f)
      x = normcdfinv((double) u);
    else {
      if (lam>400.0f) {
        int n = 1 << (tid & 7); 
        lam   = 1.0f * (float) n;
      }
      x = poissinv((double) u, (double) lam);
    }

// needed to prevent compiler discarding everything
    if (x==-999.0f) printf("negative x\n");
  }
}



//
// main code
//

int main(int argc, char **argv) {
  float lam;
  int   N, nblocks, nthreads, Count=6; 

// CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // set number of blocks, and threads per block

  N = (1<<24);
  nthreads = 256;
  nblocks  = N / nthreads;

  // execute kernels

  for (int pass=0; pass<2; pass++) {
    if (pass==0)
      printf("\nsingle precision performance tests \n");
    else
      printf("\ndouble precision performance tests \n");
    printf("---------------------------------- \n");
    printf("  lambda   execution time   samples/sec \n");

    lam = 0.125f;

    for (int count=0; count<=Count; count++) {
      lam = lam*4.0f;

      hipEventRecord(start);

      if (pass==0)
        poissinvf_test<<<nblocks,nthreads>>>(N, lam);
      else
        poissinv_test<<<nblocks,nthreads>>>(N, lam);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milli, start, stop);

// factor 10 due to repeat in test routines
// factor 1e3 due to timing in milliseconds

      if (lam>0.5f) {        // skip first one for more accurate timing
        if (count==Count)
          printf("\n normcdfinv  %9.4f     %10.3g \n",
                     milli, float(N)*10.0*1e3/milli);
        else if (count==Count-1)
          printf("   mixed     %9.4f     %10.3g \n",
                     milli, float(N)*10.0*1e3/milli);
        else
          printf("   %4g      %9.4f     %10.3g \n",
                lam, milli, float(N)*10.0*1e3/milli);
      }
    }

  }

// CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
  return 0;
}
