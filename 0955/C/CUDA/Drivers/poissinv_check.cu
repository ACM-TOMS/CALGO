#include "hip/hip_runtime.h"
//
// standard header files
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//
// CUDA header file
//

#include <hip/hip_runtime.h>

//
// my header file
//

#include "poissinv_cuda.h"

//
// function prototype for quad precision evaluation of Poisson CDF
//

extern "C" void poissinv_quad(int, float, float*, float*, double*, double*);

//
// CUDA kernels
//

__global__ void poissinvf_bisection( int N, float lam,
                                     float *ulo_d, float *uhi_d ) {

  float x, xt, u_lo, u_hi, u_mid;
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < N) {
    u_hi  = 1.0f;
    u_lo  = 0.0f;
    u_mid = 0.5f*(u_hi + u_lo);
    xt    = (float) tid;

    while (u_mid>u_lo & u_mid<u_hi) {
      x = poissinvf(u_mid, lam);

      if (x>xt)
        u_hi = u_mid;
      else
        u_lo = u_mid;

      u_mid = 0.5f*(u_hi + u_lo);
    }
    ulo_d[tid] = u_lo;
    uhi_d[tid] = u_hi;
  }
}


__global__ void poissinv_bisection( int N, double lam,
                                    double *ulo_d, double *uhi_d ) {

  double x, xt, u_lo, u_hi, u_mid;
  int   tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < N) {
    u_hi  = 1.0;
    u_lo  = 0.0;
    u_mid = 0.5*(u_hi + u_lo);
    xt    = (double) tid;

    while (u_mid>u_lo & u_mid<u_hi) {
      x = poissinv(u_mid, lam);

      if (x>xt)
        u_hi = u_mid;
      else
        u_lo = u_mid;

      u_mid = 0.5*(u_hi + u_lo);
    }
    ulo_d[tid] = u_lo;
    uhi_d[tid] = u_hi;
  }
}


//////////////////////////////////////////////////
// main code
//////////////////////////////////////////////////

int main(int argc, char **argv) {

  float   lam;
  float  *ulo_h, *uhi_h, *ulo_ex, *uhi_ex;
  float  *ulo_d, *uhi_d;

  double *Ulo_h, *Uhi_h, *Ulo_ex, *Uhi_ex;
  double *Ulo_d, *Uhi_d;

  double err1;
  int    N, nblocks, nthreads; 

  // allocate memory

  int Nmax = 2000050;   // big enough for lambda up to 10^6

  ulo_ex = (float *)malloc(Nmax*sizeof(float));
  uhi_ex = (float *)malloc(Nmax*sizeof(float));
  Ulo_ex = (double *)malloc(Nmax*sizeof(double));
  Uhi_ex = (double *)malloc(Nmax*sizeof(double));

  ulo_h = (float *)malloc(Nmax*sizeof(float));
  uhi_h = (float *)malloc(Nmax*sizeof(float));
  hipMalloc((void **)&ulo_d, Nmax*sizeof(float));
  hipMalloc((void **)&uhi_d, Nmax*sizeof(float));

  Ulo_h = (double *)malloc(Nmax*sizeof(double));
  Uhi_h = (double *)malloc(Nmax*sizeof(double));
  hipMalloc((void **)&Ulo_d, Nmax*sizeof(double));
  hipMalloc((void **)&Uhi_d, Nmax*sizeof(double));

  // set values to test

  printf("       lam     SP_err      DP_err \n");

  lam = 0.5f;

  for (int count=0; count<20; count++) {
    lam = 2.0f*lam;
    N   = 50 + (int) (2*lam);
    if (N>Nmax) exit(1);

    // set number of blocks, and threads per block

    nthreads = 256;
    nblocks  = (N-1)/nthreads + 1;

//////////////////////////////////////////////////
// compute reference solution in quad precision
//////////////////////////////////////////////////

    poissinv_quad(N, lam, ulo_ex, uhi_ex, Ulo_ex, Uhi_ex);

//////////////////////////////////////////////////
// first do tests in single precision
//////////////////////////////////////////////////

    poissinvf_bisection<<<nblocks,nthreads>>>(N, lam, ulo_d, uhi_d);

    hipMemcpy(ulo_h,ulo_d, N*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(uhi_h,uhi_d, N*sizeof(float),hipMemcpyDeviceToHost);
    err1 = 0.0;

    for (int n=0; n<N; n++) {
      err1 += 0.5*fabs( (ulo_ex[n]-ulo_h[n])
                      + (uhi_ex[n]-uhi_h[n]));

      if (n>0) {
        if ( uhi_h[n] <= ulo_ex[n-1] ) {
          printf(" error: n = %d, uhi_h[n] = %20.16g, ulo_ex[n-1] = %20.16g \n",
                          n,      uhi_h[n],           ulo_ex[n-1]);
          exit(1);
        }
        if ( uhi_ex[n] <= ulo_h[n-1] ) {
          printf(" error: n = %d, uhi_ex[n] = %20.16g, ulo_h[n-1] = %20.16g \n",
                          n,      uhi_ex[n],           ulo_h[n-1]);
          exit(1);
        }
      }
    }

    printf("%10.4g   %9.3g   ",lam,err1);

//////////////////////////////////////////////////
// now re-do tests in double precision
//////////////////////////////////////////////////

    poissinv_bisection<<<nblocks,nthreads>>>(N, lam, Ulo_d, Uhi_d);

    hipMemcpy(Ulo_h,Ulo_d, N*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(Uhi_h,Uhi_d, N*sizeof(double),hipMemcpyDeviceToHost);

    err1 = 0.0;

    for (int n=0; n<N; n++) {
      err1 += 0.5*fabs( (Ulo_ex[n]-Ulo_h[n]) + (Uhi_ex[n]-Uhi_h[n]));

      if (n>0) {
        if ( Uhi_h[n] <= Ulo_ex[n-1] ) {
          printf(" error: n = %d, Uhi_h[n] = %20.16g, Ulo_ex[n-1] = %20.16g \n",
                          n,      Uhi_h[n],           Ulo_ex[n-1]);
          exit(1);
        }
        if ( Uhi_ex[n] <= Ulo_h[n-1] ) {
          printf(" error: n = %d, Uhi_ex[n] = %20.16g, Ulo_h[n-1] = %20.16g \n",
                          n,      Uhi_ex[n],           Ulo_h[n-1]);
          exit(1);
        }
      }
    }

    printf("%9.3g \n",err1);
  }

// free memory 

  hipFree(Ulo_d);
  hipFree(Uhi_d);
  free(Ulo_h);
  free(Uhi_h);

  hipFree(ulo_d);
  hipFree(uhi_d);
  free(ulo_h);
  free(uhi_h);

  free(ulo_ex);
  free(uhi_ex);
  free(Ulo_ex);
  free(Uhi_ex);

// CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
