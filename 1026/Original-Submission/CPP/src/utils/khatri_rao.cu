#ifndef CP_CALS_KHATRI_RAO_CU
#define CP_CALS_KHATRI_RAO_CU

// #include "stdio.h"

namespace cals
{
  namespace mttkrp
  {
    __global__ void khatri_rao_cuda_kernel(double const *const A,
                                           double const *const B,
                                           unsigned int IA,
                                           unsigned int IB,
                                           unsigned int JK,
                                           double *const K)
    {
      const unsigned int n_threads = blockDim.x * gridDim.x;
      const unsigned int IAB = IA * IB;
      const unsigned int n_elements = IAB * JK;

      //const unsigned int work_per_thread = ceil((float)n_elements / (float)n_threads);
      //const unsigned int thread_idx = blockIdx.x * blockDim.x * work_per_thread + threadIdx.x * work_per_thread;
      const unsigned int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

      //for (auto i = thread_idx; i < thread_idx + work_per_thread; i ++)
      for (auto i = thread_idx; i < n_elements; i += n_threads)
      {
        //if (i < n_elements)
        //{
        const unsigned int j = i / IAB;
        const unsigned modulo = i % IAB;
        const unsigned int iA = modulo / IB;
        const unsigned int iB = modulo % IB;

        K[i] = A[iA + j * IA] * B[iB + j * IB];
        //}
      }
    }

    void khatri_rao_cuda(double *A,
                         double *B,
                         unsigned int IA,
                         unsigned int IB,
                         unsigned int JK,
                         double *K)
    {
      dim3 block_dim;
      dim3 grid_dim;
      hipOccupancyMaxPotentialBlockSize(reinterpret_cast<int *>(&grid_dim),
                                         reinterpret_cast<int *>(&block_dim),
                                         khatri_rao_cuda_kernel);
      //printf("Blocks: %d %d %d \n", block_dim.x, block_dim.y, block_dim.z);
      //printf("Grids : %d %d %d \n", grid_dim.x, grid_dim.y, grid_dim.z);
      khatri_rao_cuda_kernel<<<grid_dim, block_dim>>>(A, B, IA, IB, JK, K);
    }
  }
}
#endif //CP_CALS_KHATRI_RAO_CU
