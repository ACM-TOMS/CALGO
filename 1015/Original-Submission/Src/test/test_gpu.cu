#include "hip/hip_runtime.h"
#define LAP_CUDA
// required for multiple devices
#define LAP_CUDA_OPENMP
#define LAP_QUIET
//#define LAP_DISPLAY_EVALUATED
//#define LAP_DEBUG
//#define LAP_NO_MEM_DEBUG
//#define LAP_ROWS_SCANNED
// should only be enabled for testing purposes
//#define LAP_CUDA_ALLOW_WDDM
//#define LAP_CUDA_COMPARE_CPU
#define LAP_MINIMIZE_V

//#define RANDOM_SEED 1234

#include "../lap.h"

#include <random>
#include <string>
#include "test_options.h"
#include "image.h"
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

template <class C> void testRandom(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testSanity(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testSanityCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testGeometric(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testGeometricCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testRandomLowRank(long long min_tab, long long max_tab, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testRandomLowRankCached(long long min_cached, long long max_cached, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testImages(std::vector<std::string> &images, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);
template <class C> void testInteger(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent);


int main(int argc, char* argv[])
{
	Options opt;
	int r = opt.parseOptions(argc, argv);
	if (r != 0) return r;

	if (opt.use_double)
	{
		if (opt.use_single)
		{
			if (opt.run_sanity) testSanity<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_sanity_cached) testSanityCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random) testRandom<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random_low_rank) testRandomLowRank<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, true, std::string("double"), opt.devices, opt.silent);
			if (opt.images.size() > 1) testImages<double>(opt.images, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
		}
		if (opt.use_epsilon)
		{
			if (opt.run_sanity) testSanity<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_sanity_cached) testSanityCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random) testRandom<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random_low_rank) testRandomLowRank<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint) testGeometric<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, true, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, false, std::string("double"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<double>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, true, std::string("double"), opt.devices, opt.silent);
			if (opt.images.size() > 1) testImages<double>(opt.images, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
		}
	}
	if (opt.use_float)
	{
		if (opt.use_single)
		{
			if (opt.run_sanity) testSanity<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_sanity_cached) testSanityCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random) testRandom<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random_low_rank) testRandomLowRank<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, false, true, std::string("float"), opt.devices, opt.silent);
			if (opt.images.size() > 1) testImages<float>(opt.images, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
		}
		if (opt.use_epsilon)
		{
			if (opt.run_sanity) testSanity<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_sanity_cached) testSanityCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random) testRandom<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random_low_rank) testRandomLowRank<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_random_low_rank_cached) testRandomLowRankCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.lap_min_rank, opt.lap_max_rank, opt.runs, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint) testGeometric<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, true, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, false, std::string("float"), opt.devices, opt.silent);
			if (opt.run_geometric_disjoint_cached) testGeometricCached<float>(opt.lap_min_cached, opt.lap_max_cached, opt.lap_max_memory, opt.runs, true, true, std::string("float"), opt.devices, opt.silent);
			if (opt.images.size() > 1) testImages<float>(opt.images, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
		}
	}
	if (opt.run_integer)
	{
		if (opt.use_double)
		{
			if (opt.use_single) testInteger<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("double"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<double>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("double"), opt.devices, opt.silent);
		}
		if (opt.use_float)
		{
			if (opt.use_single) testInteger<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("float"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<float>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("float"), opt.devices, opt.silent);
		}
		if (opt.use_int)
		{
			if (opt.use_single) testInteger<int>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("int"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<int>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("int"), opt.devices, opt.silent);
		}
		if (opt.use_long)
		{
			if (opt.use_single) testInteger<long long>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, false, std::string("long long"), opt.devices, opt.silent);
			if (opt.use_epsilon) testInteger<long long>(opt.lap_min_tab, opt.lap_max_tab, opt.lap_max_memory, opt.runs, true, std::string("long long"), opt.devices, opt.silent);
		}
	}

#ifndef LAP_QUIET
	lap::allocationLogger.destroy();
#endif

	checkCudaErrors(hipProfilerStop());
	return 0;
}

template <class SC, class TC, class CF, class STATE, class TP>
void solveCachingCUDA(TP &start_time, int N1, int N2, CF &get_cost, STATE *state, lap::cuda::Worksharing &ws, long long max_memory, int *rowsol, bool epsilon)
{
	int devices = (int)ws.device.size();

	lap::cuda::SimpleCostFunction<TC, CF, STATE> costFunction(get_cost, state, devices);

	// different cache size, so always use SLRU
	lap::cuda::CachingIterator<SC, TC, decltype(costFunction), lap::CacheSLRU> iterator(N1, N2, max_memory / sizeof(TC), costFunction, ws);

	// pre-load cache
	for (int t = 0; t < devices; t++)
	{
		int rows = std::min(N1, iterator.getCache(t).getEntries());
		for (int i = 0; i < rows; i++)
		{
			int idx;
			iterator.getCache(t).find(idx, i);
		}
		checkCudaErrors(hipSetDevice(ws.device[t]));
		iterator.fillRows(t, rows);
	}
	for (int t = 0; t < devices; t++)
	{
		checkCudaErrors(hipSetDevice(ws.device[t]));
		checkCudaErrors(hipDeviceSynchronize());
	}

	lap::displayTime(start_time, "setup complete", std::cout);

	lap::cuda::solve<SC, TC>(N1, N2, costFunction, iterator, rowsol, epsilon);

	std::stringstream ss;
	ss << "cost = " << std::setprecision(std::numeric_limits<SC>::max_digits10) << lap::cuda::cost<SC, TC>(N1, N2, costFunction, rowsol, ws.stream[0]);
	lap::displayTime(start_time, ss.str().c_str(), std::cout);
}

template <class SC, class TC, class CF, class STATE, class TP>
void solveDirectCUDA(TP& start_time, int N1, int N2, CF& get_cost, STATE* state, lap::cuda::Worksharing& ws, long long max_memory, int* rowsol, bool epsilon)
{
	int devices = (int)ws.device.size();

	lap::cuda::SimpleCostFunction<TC, CF, STATE> costFunction(get_cost, state, devices);
	lap::cuda::GPUTableCost<TC> costMatrix(N1, N2, costFunction, ws);

	lap::cuda::DirectIterator<SC, TC, decltype(costMatrix)> iterator(N1, N2, costMatrix, ws);

	lap::displayTime(start_time, "setup complete", std::cout);

	lap::cuda::solve<SC, TC>(N1, N2, costMatrix, iterator, rowsol, epsilon);

	std::stringstream ss;
	ss << "cost = " << std::setprecision(std::numeric_limits<SC>::max_digits10) << lap::cuda::cost<SC, TC>(N1, N2, costFunction, rowsol, ws.stream[0]);
	lap::displayTime(start_time, ss.str().c_str(), std::cout);
}

template <class SC, class TC, class CF, class TP>
void solveCachingTableCUDA(TP &start_time, int N1, int N2, CF &get_cost_cpu, lap::cuda::Worksharing &ws, long long max_memory, int *rowsol, bool epsilon, bool sequential, bool pinned)
{
	lap::cuda::CpuCostFunction<TC, decltype(get_cost_cpu)> cpuCostFunction(get_cost_cpu, sequential);
	lap::cuda::CPUTableCost<TC> costMatrix(N1, N2, cpuCostFunction, ws, pinned);

	int devices = (int)ws.device.size();

	// different cache size, so always use SLRU
	lap::cuda::CachingIterator<SC, TC, decltype(costMatrix), lap::CacheSLRU> iterator(N1, N2, max_memory / sizeof(TC), costMatrix, ws);

	// pre-load cache
	for (int t = 0; t < devices; t++)
	{
		int rows = std::min(N1, iterator.getCache(t).getEntries());
		for (int i = 0; i < rows; i++)
		{
			int idx;
			iterator.getCache(t).find(idx, i);
		}
		checkCudaErrors(hipSetDevice(ws.device[t]));
		iterator.fillRows(t, rows);
	}
	for (int t = 0; t < devices; t++)
	{
		checkCudaErrors(hipSetDevice(ws.device[t]));
		checkCudaErrors(hipDeviceSynchronize());
	}

	lap::displayTime(start_time, "setup complete", std::cout);

	lap::cuda::solve<SC, TC>(N1, N2, costMatrix, iterator, rowsol, epsilon);

	std::stringstream ss;
	ss << "cost = " << std::setprecision(std::numeric_limits<SC>::max_digits10) << lap::cost<SC>(N1, N2, costMatrix, rowsol);
	lap::displayTime(start_time, ss.str().c_str(), std::cout);
}

template <class SC, class TC, class CF, class TP>
void solveDirectTableCUDA(TP& start_time, int N1, int N2, CF& get_cost_cpu, lap::cuda::Worksharing& ws, long long max_memory, int* rowsol, bool epsilon, bool sequential, bool pinned)
{
	lap::cuda::CpuCostFunction<TC, decltype(get_cost_cpu)> cpuCostFunction(get_cost_cpu, sequential);
	lap::cuda::CPUTableCost<TC> hostMatrix(N1, N2, cpuCostFunction, ws, pinned);
	lap::cuda::GPUTableCost<TC> costMatrix(N1, N2, hostMatrix, ws);

	int devices = (int)ws.device.size();

	lap::cuda::DirectIterator<SC, TC, decltype(costMatrix)> iterator(N1, N2, costMatrix, ws);

	lap::displayTime(start_time, "setup complete", std::cout);

	lap::cuda::solve<SC, TC>(N1, N2, costMatrix, iterator, rowsol, epsilon);

	std::stringstream ss;
	ss << "cost = " << std::setprecision(std::numeric_limits<SC>::max_digits10) << lap::cost<SC>(N1, N2, hostMatrix, rowsol);
	lap::displayTime(start_time, ss.str().c_str(), std::cout);
}

template <class SC, class TC, class CF, class STATE, class TP>
void solveCUDA(TP& start_time, int N1, int N2, CF& get_cost_cpu, STATE* state, lap::cuda::Worksharing& ws, long long max_memory, int* rowsol, bool epsilon, bool silent)
{
	bool useTable = true;
	int devices = (int)ws.device.size();
	for (int t = 0; t < devices; t++)
	{
		long long required = (long long)N1 * (long long)(ws.part[t].second - ws.part[t].first) * sizeof(TC);
		if (required > max_memory) useTable = false;
	}
	if (useTable)
	{
		if (!silent) lap::displayTime(start_time, "Solver using GPU table", std::cout);
		solveDirectCUDA<SC, TC, CF, STATE, TP>(start_time, N1, N2, get_cost_cpu, state, ws, max_memory, rowsol, epsilon);
	}
	else
	{
		if (!silent) lap::displayTime(start_time, "Solver using GPU caching", std::cout);
		solveCachingCUDA<SC, TC, CF, STATE, TP>(start_time, N1, N2, get_cost_cpu, state, ws, max_memory, rowsol, epsilon);
	}
}

template <class SC, class TC, class CF, class TP>
void solveTableCUDA(TP& start_time, int N1, int N2, CF& get_cost_cpu, lap::cuda::Worksharing& ws, long long max_memory, int* rowsol, bool epsilon, bool sequential, bool pinned, bool silent)
{
	bool useTable = true;
	int devices = (int)ws.device.size();
	for (int t = 0; t < devices; t++)
	{
		long long required = (long long)N1 * (long long)(ws.part[t].second - ws.part[t].first) * sizeof(TC);
		if (required > max_memory) useTable = false;
	}
	if (useTable)
	{
		if (!silent) lap::displayTime(start_time, "Solver using GPU table", std::cout);
		solveDirectTableCUDA<SC, TC, CF, TP>(start_time, N1, N2, get_cost_cpu, ws, max_memory, rowsol, epsilon, sequential, pinned);
	}
	else
	{
		if (!silent)
		{
			if (pinned) lap::displayTime(start_time, "Solver using GPU caching of pinned CPU table", std::cout);
			else lap::displayTime(start_time, "Solver using GPU caching of pageable CPU table", std::cout);
		}
		solveCachingTableCUDA<SC, TC, CF, TP>(start_time, N1, N2, get_cost_cpu, ws, max_memory, rowsol, epsilon, sequential, pinned);
	}
}

// needs to be declared outside of a function
template <class C>
struct GeometricState
{
	C *tab_s;
	C *tab_t;
};

template <class C>
void testGeometricCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			int max_devices = 1;
			while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

			std::cout << "Geometric";
			if (disjoint) std::cout << " Disjoint";
			std::cout << " R^2<" << name_C << "> " << N << "x" << N << " (" << (double)max_memory / 1073741824.0 << "GB / GPU)";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			auto start_time = std::chrono::high_resolution_clock::now();

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
			std::mt19937_64 generator(RANDOM_SEED);
#else
			std::random_device rd;
			std::mt19937_64 generator(rd());
#endif

			C *tab_s = new C[2 * N];
			C *tab_t = new C[2 * N];
			for (int i = 0; i < N; i++)
			{
				tab_s[i] = distribution(generator);
				tab_t[i] = distribution(generator);
				tab_s[i + N] = distribution(generator);
				tab_t[i + N] = distribution(generator);
			}

			// order of coordinates is different, first all x then all y
			if (disjoint)
			{
				for (int i = 0; i < N; i++)
				{
					if ((i << 1) < N)
					{
						tab_t[i] += C(1.0);
					}
					else
					{
						tab_s[i] += C(1.0);
						tab_s[i + N] += C(1.0);
						tab_t[i + N] += C(1.0);
					}
				}
			}

			lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);
			int num_enabled = (int)ws.device.size();

			typedef GeometricState<C> State;

			State *d_state = new State[num_enabled];

			for (int i = 0; i < num_enabled; i++)
			{
				d_state[i].tab_s = 0;
				d_state[i].tab_t = 0;
			}

			for (int i = 0; i < num_enabled; i++)
			{
				checkCudaErrors(hipSetDevice(ws.device[i]));
				lapAllocDevice(d_state[i].tab_s, 2 * N, __FILE__, __LINE__);
				lapAllocDevice(d_state[i].tab_t, 2 * N, __FILE__, __LINE__);
				checkCudaErrors(hipMemcpy(d_state[i].tab_s, tab_s, 2 * N * sizeof(C), hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(d_state[i].tab_t, tab_t, 2 * N * sizeof(C), hipMemcpyHostToDevice));
			}

			int *rowsol = new int[N];

			// cost function
			auto get_cost = [N] __device__(int x, int y, State &state)
			{
				float d0 = state.tab_s[x] - state.tab_t[y];
				float d1 = state.tab_s[x + N] - state.tab_t[y + N];
				return d0 * d0 + d1 * d1;
			};

			solveCUDA<C, C>(start_time, N, N, get_cost, d_state, ws, max_memory, rowsol, epsilon, silent);

			for (int i = 0; i < num_enabled; i++)
			{
				checkCudaErrors(hipSetDevice(ws.device[i]));
				lapFreeDevice(d_state[i].tab_s);
				lapFreeDevice(d_state[i].tab_t);
			}

			delete[] rowsol;
			delete[] tab_s;
			delete[] tab_t;
			delete[] d_state;
		}
	}
}

template <class C, class GETCOST, class STATE>
__global__
void getGTCost_sanity_kernel(C *cost, GETCOST getcost, STATE state, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= N) return;

	cost[x] = getcost(x, x, state);
}

// needs to be declared outside of a function
template <class C>
struct SanityState
{
	C *vec;
};

template <class C>
void testSanityCached(long long min_cached, long long max_cached, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			int max_devices = 1;
			while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

			std::cout << "Sanity<" << name_C << "> " << N << "x" << N << " (" << (double)max_memory / 1073741824.0 << "GB / GPU)";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			auto start_time = std::chrono::high_resolution_clock::now();

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
			std::mt19937_64 generator(RANDOM_SEED);
#else
			std::random_device rd;
			std::mt19937_64 generator(rd());
#endif

			C *vec = new C[N << 1];

			for (long long i = 0; i < N << 1; i++) vec[i] = distribution(generator);

			lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);
			int num_enabled = (int)ws.device.size();

			typedef SanityState<C> State;
			State *d_state = new State[num_enabled];

			for (int i = 0; i < num_enabled; i++)
			{
				d_state[i].vec = 0;
			}

			for (int i = 0; i < num_enabled; i++)
			{
				checkCudaErrors(hipSetDevice(ws.device[i]));
				lapAllocDevice(d_state[i].vec, 2 * N, __FILE__, __LINE__);
				checkCudaErrors(hipMemcpy(d_state[i].vec, vec, 2 * N * sizeof(C), hipMemcpyHostToDevice));
			}

			int *rowsol = new int[N];

			// cost function
			auto get_cost = [N] __device__(int x, int y, State &state)
			{
				C r = state.vec[x] + state.vec[y + N];
				if (x != y) r += C(0.1);

				return r;
			};

			solveCUDA<C, C>(start_time, N, N, get_cost, d_state, ws, max_memory, rowsol, epsilon, silent);

			bool passed = true;
			for (long long i = 0; (passed) && (i < N); i++)
			{
				passed &= (rowsol[i] == i);
			}
			std::stringstream ss;
			if (passed) ss << "test passed: ";
			else ss << "test failed: ";
			{
				// set device back to 0
				checkCudaErrors(hipSetDevice(ws.device[0]));
				C my_cost(0);
				C *row = new C[N];
				// calculate costs directly
				{
					C *d_row;
					lapAllocDevice(d_row, N, __FILE__, __LINE__);
					dim3 block_size, grid_size;
					block_size.x = 256;
					grid_size.x = (N + block_size.x - 1) / block_size.x;
					getGTCost_sanity_kernel<<<grid_size, block_size>>>(d_row, get_cost, d_state[0], N);
					checkCudaErrors(hipMemcpy(row, d_row, N * sizeof(C), hipMemcpyDeviceToHost));
					lapFreeDevice(d_row);
				}
				for (int i = 0; i < N; i++) my_cost += row[i];
				delete[] row;
				ss << "ground truth cost = " << std::setprecision(std::numeric_limits<C>::max_digits10) << my_cost;
			}
			lap::displayTime(start_time, ss.str().c_str(), std::cout);

			for (int i = 0; i < num_enabled; i++)
			{
				checkCudaErrors(hipSetDevice(ws.device[i]));
				lapFreeDevice(d_state[i].vec);
			}

			delete[] rowsol;
			delete[] vec;
			delete[] d_state;
		}
	}
}

template <class C>
void testRandomLowRankCached(long long min_cached, long long max_cached, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	for (long long rank = min_rank; rank <= max_rank; rank <<= 1)
	{
		for (long long NN = min_cached * min_cached; NN <= max_cached * max_cached; NN <<= 1)
		{
			for (int r = 0; r < runs; r++)
			{
				int N = (int)floor(sqrt((double)NN));

				int max_devices = 1;
				while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

				std::cout << "RandomLowRank<" << name_C << "> " << N << "x" << N << " (" << (double)max_memory / 1073741824.0 << "GB / GPU)";
				if (epsilon) std::cout << " with epsilon scaling";
				std::cout << std::endl;

				auto start_time = std::chrono::high_resolution_clock::now();

				std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
				std::mt19937_64 generator(RANDOM_SEED);
#else
				std::random_device rd;
				std::mt19937_64 generator(rd());
#endif

				// The following matrix will have at most the seletcted rank.
				C *vec = new C[N * rank];
				for (long long i = 0; i < rank; i++)
				{
					for (long long j = 0; j < N; j++) vec[i * N + j] = distribution(generator);
				}

				lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);
				int num_enabled = (int)ws.device.size();

				typedef SanityState<C> State;
				State *d_state = new State[num_enabled];

				for (int i = 0; i < num_enabled; i++)
				{
					d_state[i].vec = 0;
				}

				for (int i = 0; i < num_enabled; i++)
				{
					checkCudaErrors(hipSetDevice(ws.device[i]));
					lapAllocDevice(d_state[i].vec, N * rank, __FILE__, __LINE__);
					checkCudaErrors(hipMemcpy(d_state[i].vec, vec, N * rank * sizeof(C), hipMemcpyHostToDevice));
				}

				int *rowsol = new int[N];

				// cost function
				auto get_cost = [rank, N] __device__(int x, int y, State &state)
				{
					C sum(0);
#pragma unroll(8)
					for (long long k = 0; k < rank; k++)
					{
						sum += state.vec[k * N + x] * state.vec[k * N + y];
					}
					sum /= C(rank);

					return sum;
				};

				solveCUDA<C, C>(start_time, N, N, get_cost, d_state, ws, max_memory, rowsol, epsilon, silent);

				for (int i = 0; i < num_enabled; i++)
				{
					checkCudaErrors(hipSetDevice(ws.device[i]));
					lapFreeDevice(d_state[i].vec);
				}

				delete[] rowsol;
				delete[] vec;
				delete[] d_state;
			}
		}
	}
}

template <class C>
void testInteger(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	// random costs (directly supply cost matrix)
	//for (int range = 0; range < 3; range++)
	int range = 2;
	{
		for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
		{
			for (int r = 0; r < runs; r++)
			{
				int N = (int)floor(sqrt((double)NN));

				int max_devices = 1;
				while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

				std::cout << "Integer";
				std::cout << "<" << name_C << " ";
				if (range == 0) std::cout << "1/10n";
				else if (range == 1) std::cout << "n";
				else std::cout << "10n";
				std::cout << "> " << N << "x" << N << " table";
				if (epsilon) std::cout << " with epsilon scaling";
				std::cout << std::endl;

				int n;
				if (range == 0) n = N / 10;
				else if (range == 1) n = N;
				else n = 10 * N;
				std::uniform_int_distribution<int> distribution(0, n);
#ifdef RANDOM_SEED
				std::mt19937_64 generator(RANDOM_SEED);
#else
				std::random_device rd;
				std::mt19937_64 generator(rd());
#endif

				auto start_time = std::chrono::high_resolution_clock::now();

				auto get_cost = [&distribution, &generator](int x, int y) -> int
				{
					return distribution(generator);
				};

				lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);

				int *rowsol = new int[N];

				solveTableCUDA<C, int>(start_time, N, N, get_cost, ws, max_memory, rowsol, epsilon, true, N < max_tab, silent);

				delete[] rowsol;
			}
		}
	}
}

template <class C> void testRandom(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	// random costs (directly supply cost matrix)
	for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			int max_devices = 1;
			while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

			std::cout << "Random";
			std::cout << "<" << name_C << "> " << N << "x" << N << " table";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
			std::mt19937_64 generator(RANDOM_SEED);
#else
			std::random_device rd;
			std::mt19937_64 generator(rd());
#endif

			auto start_time = std::chrono::high_resolution_clock::now();

			int *rowsol = new int[N];

			auto get_cost = [&distribution, &generator](int x, int y) -> C
			{
				return distribution(generator);
			};

			lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);

			solveTableCUDA<C, C>(start_time, N, N, get_cost, ws, max_memory, rowsol, epsilon, true, N < max_tab, silent);

			delete[] rowsol;
		}
	}
}

template <class C> void testSanity(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	// random costs (directly supply cost matrix)
	for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			int max_devices = 1;
			while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

			std::cout << "Sanity";
			std::cout << "<" << name_C << "> " << N << "x" << N << " table";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
			std::mt19937_64 generator(RANDOM_SEED);
#else
			std::random_device rd;
			std::mt19937_64 generator(rd());
#endif

			auto start_time = std::chrono::high_resolution_clock::now();

			int *rowsol = new int[N];

			C *vec = new C[N << 1];

			for (long long i = 0; i < N << 1; i++) vec[i] = distribution(generator);

			// cost functions
			auto get_cost = [&vec, &N](int x, int y) -> C
			{
				C r = vec[x] + vec[y + N];
				if (x == y) return r;
				else return r + C(0.1);
			};


			lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);

			solveTableCUDA<C, C>(start_time, N, N, get_cost, ws, max_memory, rowsol, epsilon, false, N < max_tab, silent);

			bool passed = true;
			for (long long i = 0; (passed) && (i < N); i++)
			{
				passed &= (rowsol[i] == i);
			}
			std::stringstream ss;
			if (passed) ss << "test passed: ";
			else ss << "test failed: ";
			C real_cost(0);
			for (int i = 0; i < N; i++) real_cost += get_cost(i, i);
			ss << "ground truth cost = " << std::setprecision(std::numeric_limits<C>::max_digits10) << real_cost;
			lap::displayTime(start_time, ss.str().c_str(), std::cout);

			delete[] vec;
			delete[] rowsol;
		}
	}
}

template <class C> void testGeometric(long long min_tab, long long max_tab, long long max_memory, int runs, bool epsilon, bool disjoint, std::string name_C, std::vector<int> &devs, bool silent)
{
	// geometric costs in R^2 (supply function for calculating cost matrix)
	for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
	{
		for (int r = 0; r < runs; r++)
		{
			int N = (int)floor(sqrt((double)NN));

			int max_devices = 1;
			while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

			std::cout << "Geometric";
			if (disjoint) std::cout << " Disjoint";
			std::cout << " R^2<" << name_C << "> " << N << "x" << N << " table";
			if (epsilon) std::cout << " with epsilon scaling";
			std::cout << std::endl;

			auto start_time = std::chrono::high_resolution_clock::now();

			std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
			std::mt19937_64 generator(RANDOM_SEED);
#else
			std::random_device rd;
			std::mt19937_64 generator(rd());
#endif

			C *tab_s = new C[2 * N];
			C *tab_t = new C[2 * N];

			for (int i = 0; i < 2 * N; i++)
			{
				tab_s[i] = distribution(generator);
				tab_t[i] = distribution(generator);
			}

			if (disjoint)
			{
				for (int i = 0; i < 2 * N; i += 2)
				{
					if (i < N)
					{
						tab_t[i] += C(1);
					}
					else
					{
						tab_s[i] += C(1);
						tab_s[i + 1] += C(1);
						tab_t[i + 1] += C(1);
					}
				}
			}

			// cost function
			auto get_cost = [&tab_s, &tab_t](int x, int y) -> C
			{
				int xx = x + x;
				int yy = y + y;
				C d0 = tab_s[xx] - tab_t[yy];
				C d1 = tab_s[xx + 1] - tab_t[yy + 1];
				return d0 * d0 + d1 * d1;
			};

			int *rowsol = new int[N];

			lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);

			solveTableCUDA<C, C>(start_time, N, N, get_cost, ws, max_memory, rowsol, epsilon, false, N < max_tab, silent);

			delete[] tab_s;
			delete[] tab_t;
			delete[] rowsol;
		}
	}
}

template <class C> void testRandomLowRank(long long min_tab, long long max_tab, long long max_memory, long long min_rank, long long max_rank, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	// random costs (directly supply cost matrix)
	for (long long rank = min_rank; rank <= max_rank; rank <<= 1)
	{
		for (long long NN = min_tab * min_tab; NN <= max_tab * max_tab; NN <<= 1)
		{
			for (int r = 0; r < runs; r++)
			{
				int N = (int)floor(sqrt((double)NN));

				int max_devices = 1;
				while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N * (size_t)N) max_devices++;

				std::cout << "RandomLowRank<" << name_C << "> " << N << "x" << N << " table rank = " << rank;
				if (epsilon) std::cout << " with epsilon scaling";
				std::cout << std::endl;

				auto start_time = std::chrono::high_resolution_clock::now();

				std::uniform_real_distribution<C> distribution(0.0, 1.0);
#ifdef RANDOM_SEED
				std::mt19937_64 generator(RANDOM_SEED);
#else
				std::random_device rd;
				std::mt19937_64 generator(rd());
#endif

				// The following matrix will have at most the seletcted rank.
				C *vec = new C[N * rank];
				C max_vec;
				C min_vec;
				for (long long i = 0; i < rank; i++)
				{
					for (long long j = 0; j < N; j++) vec[i * N + j] = distribution(generator);
					max_vec = vec[i * N];
					for (long long j = 1; j < N; j++) max_vec = std::max(max_vec, vec[i * N + j]);
					min_vec = vec[i * N];
					for (long long j = 1; j < N; j++) min_vec = std::min(min_vec, vec[i * N + j]);
				}

				// cost function
				auto get_cost = [&vec, &N, &rank, &max_vec](int x, int y) -> C
				{
					C sum(0);
					for (long long k = 0; k < rank; k++)
					{
						sum += vec[k * N + x] * vec[k * N + y];
					}
					return sum / C(rank);
				};

				int *rowsol = new int[N];

				lap::cuda::Worksharing ws(N, 256, devs, max_devices, silent);

				solveTableCUDA<C, C>(start_time, N, N, get_cost, ws, max_memory, rowsol, epsilon, false, N < max_tab, silent);

				delete[] vec;
				delete[] rowsol;
			}
		}
	}
}

// needs to be declared outside of a function
struct ImagesState
{
	unsigned char *c00;
	unsigned char *c01;
	unsigned char *c02;
	unsigned char *c10;
	unsigned char *c11;
	unsigned char *c12;
};

template <class C> void testImages(std::vector<std::string> &images, long long max_memory, int runs, bool epsilon, std::string name_C, std::vector<int> &devs, bool silent)
{
	std::cout << "Comparing images ";
	if (epsilon) std::cout << " with epsilon scaling";
	std::cout << std::endl;
	for (unsigned int a = 0; a < images.size() - 1; a++)
	{
		for (unsigned int b = a + 1; b < images.size(); b++)
		{
			PPMImage img_a(images[a]);
			PPMImage img_b(images[b]);
			std::cout << "Comparing image \"" << images[a] << "\" (" << img_a.width << "x" << img_a.height << ") with image \"" << images[b] << "\" (" << img_b.width << "x" << img_b.height << ")." << std::endl;
			for (int r = 0; r < runs; r++)
			{
				auto start_time = std::chrono::high_resolution_clock::now();

				int N1 = std::min(img_a.width * img_a.height, img_b.width * img_b.height);
				int N2 = std::max(img_a.width * img_a.height, img_b.width * img_b.height);

				long long max_memory_local = max_memory - (N1 * 3 + N2 + 3);

				int max_devices = 1;
				while ((size_t)max_devices * max_memory / sizeof(C) < (size_t)N1 * (size_t)N2) max_devices++;

				lap::cuda::Worksharing ws(N2, 256, devs, max_devices, silent);
				int num_devices = (int)ws.device.size();
				typedef ImagesState State;
				// make sure img[0] is at most as large as img[1]
				PPMImage *img[2];
				img[0] = new PPMImage[num_devices];
				img[1] = new PPMImage[num_devices];
				// rearrange data for GPU
				int size_a = img_a.width * img_a.height;
				unsigned char *buf_a = new unsigned char[size_a * 3];
				int size_b = img_b.width * img_b.height;
				unsigned char *buf_b = new unsigned char[size_b * 3];
				for (int y = 0; y < img_a.height; y++)
				{
					for (int x = 0; x < img_a.width; x++)
					{
						int off = x + y * img_a.width;
						buf_a[off] = img_a.raw[off * 3];
						buf_a[off + size_a] = img_a.raw[off * 3 + 1];
						buf_a[off + 2 * size_a] = img_a.raw[off * 3 + 2];
					}
				}
				for (int y = 0; y < img_b.height; y++)
				{
					for (int x = 0; x < img_b.width; x++)
					{
						int off = x + y * img_b.width;
						buf_b[off] = img_b.raw[off * 3];
						buf_b[off + size_b] = img_b.raw[off * 3 + 1];
						buf_b[off + 2 * size_b] = img_b.raw[off * 3 + 2];
					}
				}
				for (int t = 0; t < num_devices; t++)
				{
					checkCudaErrors(hipSetDevice(ws.device[t]));
					if (img_a.width * img_a.height < img_b.width * img_b.height)
					{
						img[0][t].width = img_a.width;
						img[0][t].height = img_a.height;
						img[0][t].max_val = img_a.max_val;
						lapAllocDevice(img[0][t].raw, img[0][t].width * img[0][t].height * 3, __FILE__, __LINE__);
						img[1][t].width = img_b.width;
						img[1][t].height = img_b.height;
						img[1][t].max_val = img_b.max_val;
						lapAllocDevice(img[1][t].raw, img[1][t].width * img[1][t].height * 3, __FILE__, __LINE__);

						checkCudaErrors(hipMemcpyAsync(img[0][t].raw, buf_a, img[0][t].width * img[0][t].height * 3, hipMemcpyHostToDevice));
						checkCudaErrors(hipMemcpyAsync(img[1][t].raw, buf_b, img[1][t].width * img[1][t].height * 3, hipMemcpyHostToDevice));
					}
					else
					{
						img[0][t].width = img_b.width;
						img[0][t].height = img_b.height;
						img[0][t].max_val = img_b.max_val;
						lapAllocDevice(img[0][t].raw, img[0][t].width * img[0][t].height * 3, __FILE__, __LINE__);
						img[1][t].width = img_a.width;
						img[1][t].height = img_a.height;
						img[1][t].max_val = img_a.max_val;
						lapAllocDevice(img[1][t].raw, img[1][t].width * img[1][t].height * 3, __FILE__, __LINE__);

						checkCudaErrors(hipMemcpyAsync(img[1][t].raw, buf_a, img[1][t].width * img[1][t].height * 3, hipMemcpyHostToDevice));
						checkCudaErrors(hipMemcpyAsync(img[0][t].raw, buf_b, img[0][t].width * img[0][t].height * 3, hipMemcpyHostToDevice));
					}
					checkCudaErrors(hipDeviceSynchronize());
				}
				delete[] buf_a;
				delete[] buf_b;

				// setup state and other arguments
				typedef ImagesState State;
				State *d_state = new State[num_devices];
				int w0 = img[0][0].width;
				int h0 = img[0][0].height;
				int mval0 = img[0][0].max_val;
				int w1 = img[1][0].width;
				int h1 = img[1][0].height;
				int mval1 = img[1][0].max_val;
				int size_0 = img[0][0].width * img[0][0].height;
				int size_1 = img[1][0].width * img[1][0].height;
				for (int t = 0; t < num_devices; t++)
				{
					d_state[t].c00 = img[0][t].raw;
					d_state[t].c01 = img[0][t].raw + size_0;
					d_state[t].c02 = img[0][t].raw + 2 * size_0;
					d_state[t].c10 = img[1][t].raw;
					d_state[t].c11 = img[1][t].raw + size_1;
					d_state[t].c12 = img[1][t].raw + 2 * size_1;
				}

				auto get_cost = [w0, h0, mval0, w1, h1, mval1] __device__(int x, int y, State &state)
				{
					C r = C(state.c00[x]) / C(mval0) - C(state.c10[y]) / C(mval1);
					C g = C(state.c01[x]) / C(mval0) - C(state.c11[y]) / C(mval1);
					C b = C(state.c02[x]) / C(mval0) - C(state.c12[y]) / C(mval1);
					C u = C(x % w0) / C(w0 - 1) - C(y % w1) / C(w1 - 1);
					C v = C(x / w0) / C(h0 - 1) - C(y / w1) / C(h1 - 1);
					return r * r + g * g + b * b + u * u + v * v;
				};

				int *rowsol = new int[N2];

				solveCUDA<C, C>(start_time, N1, N2, get_cost, d_state, ws, max_memory_local, rowsol, epsilon, silent);

				for (int t = 0; t < num_devices; t++)
				{
					lapFreeDevice(img[0][t].raw);
					lapFreeDevice(img[1][t].raw);
				}
				delete[] rowsol;
				delete[] img[0];
				delete[] img[1];
				delete[] d_state;
			}
		}
	}
}
